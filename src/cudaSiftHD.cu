#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Mårten Björkman aka Celebrandil //
//********************************************************//  

#include <cstdio>
#include <cstring>
#include <cmath>
#include <iostream>
#include <algorithm>

#include <cuda_sift/cudautils.h>
#include <cuda_sift/cudaImage.h>
#include <cuda_sift/cudaSift.h>
#include <cuda_sift/cudaSiftD.h>
#include <cuda_sift/cudaSiftH.h>
#include <hip/driver_types.h>


///////////////////////////////////////////////////////////////////////////////
// Kernel configuration
///////////////////////////////////////////////////////////////////////////////

__constant__ int d_MaxNumPoints;
__device__ unsigned int d_PointCounter[8*2+1];
__constant__ float d_ScaleDownKernel[5];
__constant__ float d_LowPassKernel[2*LOWPASS_R+1];
__constant__ float d_LaplaceKernel[8*12*16];

///////////////////////////////////////////////////////////////////////////////
// Lowpass filter and subsample image
///////////////////////////////////////////////////////////////////////////////
__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    __shared__ float inrow[SCALEDOWN_W+4];
    __shared__ float brow[5*(SCALEDOWN_W/2)];
    __shared__ int yRead[SCALEDOWN_H+4];
    __shared__ int yWrite[SCALEDOWN_H+4];
#define dx2 (SCALEDOWN_W/2)
    const int tx = threadIdx.x;
    const int tx0 = tx + 0*dx2;
    const int tx1 = tx + 1*dx2;
    const int tx2 = tx + 2*dx2;
    const int tx3 = tx + 3*dx2;
    const int tx4 = tx + 4*dx2;
    const int xStart = blockIdx.x*SCALEDOWN_W;
    const int yStart = blockIdx.y*SCALEDOWN_H;
    const int xWrite = xStart/2 + tx;
    const float *k = d_ScaleDownKernel;
    if (tx<SCALEDOWN_H+4) {
        int y = yStart + tx - 1;
        y = (y<0 ? 0 : y);
        y = (y>=height ? height-1 : y);
        yRead[tx] = y*pitch;
        yWrite[tx] = (yStart + tx - 4)/2 * newpitch;
    }
    __syncthreads();
    int xRead = xStart + tx - 2;
    xRead = (xRead<0 ? 0 : xRead);
    xRead = (xRead>=width ? width-1 : xRead);
    for (int dy=0;dy<SCALEDOWN_H+4;dy+=5) {
        inrow[tx] = d_Data[yRead[dy+0] + xRead];
        __syncthreads();
        if (tx<dx2)
            brow[tx0] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
        __syncthreads();
        if (tx<dx2 && dy>=4 && !(dy&1))
            d_Result[yWrite[dy+0] + xWrite] = k[2]*brow[tx2] + k[0]*(brow[tx0]+brow[tx4]) + k[1]*(brow[tx1]+brow[tx3]);
        if (dy<(SCALEDOWN_H+3)) {
            inrow[tx] = d_Data[yRead[dy+1] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx1] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && dy>=3 && (dy&1))
                d_Result[yWrite[dy+1] + xWrite] = k[2]*brow[tx3] + k[0]*(brow[tx1]+brow[tx0]) + k[1]*(brow[tx2]+brow[tx4]);
        }
        if (dy<(SCALEDOWN_H+2)) {
            inrow[tx] = d_Data[yRead[dy+2] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx2] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && dy>=2 && !(dy&1))
                d_Result[yWrite[dy+2] + xWrite] = k[2]*brow[tx4] + k[0]*(brow[tx2]+brow[tx1]) + k[1]*(brow[tx3]+brow[tx0]);
        }
        if (dy<(SCALEDOWN_H+1)) {
            inrow[tx] = d_Data[yRead[dy+3] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx3] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && dy>=1 && (dy&1))
                d_Result[yWrite[dy+3] + xWrite] = k[2]*brow[tx0] + k[0]*(brow[tx3]+brow[tx2]) + k[1]*(brow[tx4]+brow[tx1]);
        }
        if (dy<SCALEDOWN_H) {
            inrow[tx] = d_Data[yRead[dy+4] + xRead];
            __syncthreads();
            if (tx<dx2)
                brow[tx4] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
            __syncthreads();
            if (tx<dx2 && !(dy&1))
                d_Result[yWrite[dy+4] + xWrite] = k[2]*brow[tx1] + k[0]*(brow[tx4]+brow[tx3]) + k[1]*(brow[tx0]+brow[tx2]);
        }
        __syncthreads();
    }
}

__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
#define BW (SCALEUP_W/2 + 2)
#define BH (SCALEUP_H/2 + 2)
    __shared__ float buffer[BW*BH];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    if (tx<BW && ty<BH) {
        int x = min(max(blockIdx.x*(SCALEUP_W/2) + tx - 1, 0), width-1);
        int y = min(max(blockIdx.y*(SCALEUP_H/2) + ty - 1, 0), height-1);
        buffer[ty*BW + tx] = d_Data[y*pitch + x];
    }
    __syncthreads();
    int x = blockIdx.x*SCALEUP_W + tx;
    int y = blockIdx.y*SCALEUP_H + ty;
    if (x<2*width && y<2*height) {
        int bx = (tx + 1)/2;
        int by = (ty + 1)/2;
        int bp = by*BW + bx;
        float wx = 0.25f + (tx&1)*0.50f;
        float wy = 0.25f + (ty&1)*0.50f;
        d_Result[y*newpitch + x] = wy*(wx*buffer[bp] + (1.0f-wx)*buffer[bp+1]) +
                                   (1.0f-wy)*(wx*buffer[bp+BW] + (1.0f-wx)*buffer[bp+BW+1]);
    }
}

__global__ void ExtractSiftDescriptors(hipTextureObject_t texObj, SiftPoint *d_sift, int fstPts, float subsampling)
{
    __shared__ float gauss[16];
    __shared__ float buffer[128];
    __shared__ float sums[4];

    const int tx = threadIdx.x; // 0 -> 16
    const int ty = threadIdx.y; // 0 -> 8
    const int idx = ty*16 + tx;
    const int bx = blockIdx.x + fstPts;  // 0 -> numPts
    if (ty==0)
        gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
    buffer[idx] = 0.0f;
    __syncthreads();

    // Compute angles and gradients
    float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
    float sina = sinf(theta);           // cosa -sina
    float cosa = cosf(theta);           // sina  cosa
    float scale = 12.0f/16.0f*d_sift[bx].scale;
    float ssina = scale*sina;
    float scosa = scale*cosa;

    for (int y=ty;y<16;y+=8) {
        float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina;
        float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa;
        float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) -
                   tex2D<float>(texObj, xpos-cosa, ypos-sina);
        float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) -
                   tex2D<float>(texObj, xpos+sina, ypos-cosa);
        float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
        float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;

        int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins
        float horf = (tx - 1.5f)/4.0f - hori;
        float ihorf = 1.0f - horf;
        int veri = (y + 2)/4 - 1;
        float verf = (y - 1.5f)/4.0f - veri;
        float iverf = 1.0f - verf;
        int angi = angf;
        int angp = (angi<7 ? angi+1 : 0);
        angf -= angi;
        float iangf = 1.0f - angf;

        int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated
        int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
        int p2 = angp + hist;
        if (tx>=2) {
            float grad1 = ihorf*grad;
            if (y>=2) {   // Upper left
                float grad2 = iverf*grad1;
                atomicAdd(buffer + p1, iangf*grad2);
                atomicAdd(buffer + p2,  angf*grad2);
            }
            if (y<=13) {  // Lower left
                float grad2 = verf*grad1;
                atomicAdd(buffer + p1+32, iangf*grad2);
                atomicAdd(buffer + p2+32,  angf*grad2);
            }
        }
        if (tx<=13) {
            float grad1 = horf*grad;
            if (y>=2) {    // Upper right
                float grad2 = iverf*grad1;
                atomicAdd(buffer + p1+8, iangf*grad2);
                atomicAdd(buffer + p2+8,  angf*grad2);
            }
            if (y<=13) {   // Lower right
                float grad2 = verf*grad1;
                atomicAdd(buffer + p1+40, iangf*grad2);
                atomicAdd(buffer + p2+40,  angf*grad2);
            }
        }
    }
    __syncthreads();

    // Normalize twice and suppress peaks first time
    float sum = buffer[idx]*buffer[idx];
    for (int i=16;i>0;i/=2)
        sum += ShiftDown(sum, i);
    if ((idx&31)==0)
        sums[idx/32] = sum;
    __syncthreads();
    float tsum1 = sums[0] + sums[1] + sums[2] + sums[3];
    tsum1 = min(buffer[idx] * rsqrtf(tsum1), 0.2f);

    sum = tsum1*tsum1;
    for (int i=16;i>0;i/=2)
        sum += ShiftDown(sum, i);
    if ((idx&31)==0)
        sums[idx/32] = sum;
    __syncthreads();

    float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];
    float *desc = d_sift[bx].data;
    desc[idx] = tsum1 * rsqrtf(tsum2);
    if (idx==0) {
        d_sift[bx].xpos *= subsampling;
        d_sift[bx].ypos *= subsampling;
        d_sift[bx].scale *= subsampling;
    }
}


__global__ void ExtractSiftDescriptorsCONST(hipTextureObject_t texObj, SiftPoint *d_sift, float subsampling, int octave)
{
    __shared__ float gauss[16];
    __shared__ float buffer[128];
    __shared__ float sums[4];

    const int tx = threadIdx.x; // 0 -> 16
    const int ty = threadIdx.y; // 0 -> 8
    const int idx = ty*16 + tx;
    if (ty==0)
        gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);

    int fstPts = min(d_PointCounter[2*octave-1], d_MaxNumPoints);
    int totPts = min(d_PointCounter[2*octave+1], d_MaxNumPoints);
    //if (tx==0 && ty==0)
    //  printf("%d %d %d %d\n", octave, fstPts, min(d_PointCounter[2*octave], d_MaxNumPoints), totPts);
    for (int bx = blockIdx.x + fstPts; bx < totPts; bx += gridDim.x) {

        buffer[idx] = 0.0f;
        __syncthreads();

        // Compute angles and gradients
        float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
        float sina = sinf(theta);           // cosa -sina
        float cosa = cosf(theta);           // sina  cosa
        float scale = 12.0f/16.0f*d_sift[bx].scale;
        float ssina = scale*sina;
        float scosa = scale*cosa;

        for (int y=ty;y<16;y+=8) {
            float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina;
            float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa;
            float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) -
                       tex2D<float>(texObj, xpos-cosa, ypos-sina);
            float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) -
                       tex2D<float>(texObj, xpos+sina, ypos-cosa);
            float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
            float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;

            int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins
            float horf = (tx - 1.5f)/4.0f - hori;
            float ihorf = 1.0f - horf;
            int veri = (y + 2)/4 - 1;
            float verf = (y - 1.5f)/4.0f - veri;
            float iverf = 1.0f - verf;
            int angi = angf;
            int angp = (angi<7 ? angi+1 : 0);
            angf -= angi;
            float iangf = 1.0f - angf;

            int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated
            int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
            int p2 = angp + hist;
            if (tx>=2) {
                float grad1 = ihorf*grad;
                if (y>=2) {   // Upper left
                    float grad2 = iverf*grad1;
                    atomicAdd(buffer + p1, iangf*grad2);
                    atomicAdd(buffer + p2,  angf*grad2);
                }
                if (y<=13) {  // Lower left
                    float grad2 = verf*grad1;
                    atomicAdd(buffer + p1+32, iangf*grad2);
                    atomicAdd(buffer + p2+32,  angf*grad2);
                }
            }
            if (tx<=13) {
                float grad1 = horf*grad;
                if (y>=2) {    // Upper right
                    float grad2 = iverf*grad1;
                    atomicAdd(buffer + p1+8, iangf*grad2);
                    atomicAdd(buffer + p2+8,  angf*grad2);
                }
                if (y<=13) {   // Lower right
                    float grad2 = verf*grad1;
                    atomicAdd(buffer + p1+40, iangf*grad2);
                    atomicAdd(buffer + p2+40,  angf*grad2);
                }
            }
        }
        __syncthreads();

        // Normalize twice and suppress peaks first time
        float sum = buffer[idx]*buffer[idx];
        for (int i=16;i>0;i/=2)
            sum += ShiftDown(sum, i);
        if ((idx&31)==0)
            sums[idx/32] = sum;
        __syncthreads();
        float tsum1 = sums[0] + sums[1] + sums[2] + sums[3];
        tsum1 = min(buffer[idx] * rsqrtf(tsum1), 0.2f);

        sum = tsum1*tsum1;
        for (int i=16;i>0;i/=2)
            sum += ShiftDown(sum, i);
        if ((idx&31)==0)
            sums[idx/32] = sum;
        __syncthreads();

        float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];
        float *desc = d_sift[bx].data;
        desc[idx] = tsum1 * rsqrtf(tsum2);
        if (idx==0) {
            d_sift[bx].xpos *= subsampling;
            d_sift[bx].ypos *= subsampling;
            d_sift[bx].scale *= subsampling;
        }
        __syncthreads();
    }
}


__global__ void ExtractSiftDescriptorsOld(hipTextureObject_t texObj, SiftPoint *d_sift, int fstPts, float subsampling)
{
    __shared__ float gauss[16];
    __shared__ float buffer[128];
    __shared__ float sums[128];

    const int tx = threadIdx.x; // 0 -> 16
    const int ty = threadIdx.y; // 0 -> 8
    const int idx = ty*16 + tx;
    const int bx = blockIdx.x + fstPts;  // 0 -> numPts
    if (ty==0)
        gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
    buffer[idx] = 0.0f;
    __syncthreads();

    // Compute angles and gradients
    float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
    float sina = sinf(theta);           // cosa -sina
    float cosa = cosf(theta);           // sina  cosa
    float scale = 12.0f/16.0f*d_sift[bx].scale;
    float ssina = scale*sina;
    float scosa = scale*cosa;

    for (int y=ty;y<16;y+=8) {
        float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina;
        float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa;
        float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) -
                   tex2D<float>(texObj, xpos-cosa, ypos-sina);
        float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) -
                   tex2D<float>(texObj, xpos+sina, ypos-cosa);
        float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
        float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;

        int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins
        float horf = (tx - 1.5f)/4.0f - hori;
        float ihorf = 1.0f - horf;
        int veri = (y + 2)/4 - 1;
        float verf = (y - 1.5f)/4.0f - veri;
        float iverf = 1.0f - verf;
        int angi = angf;
        int angp = (angi<7 ? angi+1 : 0);
        angf -= angi;
        float iangf = 1.0f - angf;

        int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated
        int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
        int p2 = angp + hist;
        if (tx>=2) {
            float grad1 = ihorf*grad;
            if (y>=2) {   // Upper left
                float grad2 = iverf*grad1;
                atomicAdd(buffer + p1, iangf*grad2);
                atomicAdd(buffer + p2,  angf*grad2);
            }
            if (y<=13) {  // Lower left
                float grad2 = verf*grad1;
                atomicAdd(buffer + p1+32, iangf*grad2);
                atomicAdd(buffer + p2+32,  angf*grad2);
            }
        }
        if (tx<=13) {
            float grad1 = horf*grad;
            if (y>=2) {    // Upper right
                float grad2 = iverf*grad1;
                atomicAdd(buffer + p1+8, iangf*grad2);
                atomicAdd(buffer + p2+8,  angf*grad2);
            }
            if (y<=13) {   // Lower right
                float grad2 = verf*grad1;
                atomicAdd(buffer + p1+40, iangf*grad2);
                atomicAdd(buffer + p2+40,  angf*grad2);
            }
        }
    }
    __syncthreads();

    // Normalize twice and suppress peaks first time
    if (idx<64)
        sums[idx] = buffer[idx]*buffer[idx] + buffer[idx+64]*buffer[idx+64];
    __syncthreads();
    if (idx<32) sums[idx] = sums[idx] + sums[idx+32];
    __syncthreads();
    if (idx<16) sums[idx] = sums[idx] + sums[idx+16];
    __syncthreads();
    if (idx<8)  sums[idx] = sums[idx] + sums[idx+8];
    __syncthreads();
    if (idx<4)  sums[idx] = sums[idx] + sums[idx+4];
    __syncthreads();
    float tsum1 = sums[0] + sums[1] + sums[2] + sums[3];
    buffer[idx] = buffer[idx] * rsqrtf(tsum1);

    if (buffer[idx]>0.2f)
        buffer[idx] = 0.2f;
    __syncthreads();
    if (idx<64)
        sums[idx] = buffer[idx]*buffer[idx] + buffer[idx+64]*buffer[idx+64];
    __syncthreads();
    if (idx<32) sums[idx] = sums[idx] + sums[idx+32];
    __syncthreads();
    if (idx<16) sums[idx] = sums[idx] + sums[idx+16];
    __syncthreads();
    if (idx<8)  sums[idx] = sums[idx] + sums[idx+8];
    __syncthreads();
    if (idx<4)  sums[idx] = sums[idx] + sums[idx+4];
    __syncthreads();
    float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];

    float *desc = d_sift[bx].data;
    desc[idx] = buffer[idx] * rsqrtf(tsum2);
    if (idx==0) {
        d_sift[bx].xpos *= subsampling;
        d_sift[bx].ypos *= subsampling;
        d_sift[bx].scale *= subsampling;
    }
}


__global__ void RescalePositions(SiftPoint *d_sift, int numPts, float scale)
{
    int num = blockIdx.x*blockDim.x + threadIdx.x;
    if (num<numPts) {
        d_sift[num].xpos *= scale;
        d_sift[num].ypos *= scale;
        d_sift[num].scale *= scale;
    }
}


__global__ void ComputeOrientations(hipTextureObject_t texObj, SiftPoint *d_Sift, int fstPts)
{
    __shared__ float hist[64];
    __shared__ float gauss[11];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x + fstPts;
    float i2sigma2 = -1.0f/(4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
    if (tx<11)
        gauss[tx] = exp(i2sigma2*(tx-5)*(tx-5));
    if (tx<64)
        hist[tx] = 0.0f;
    __syncthreads();
    float xp = d_Sift[bx].xpos - 5.0f;
    float yp = d_Sift[bx].ypos - 5.0f;
    int yd = tx/11;
    int xd = tx - yd*11;
    float xf = xp + xd;
    float yf = yp + yd;
    if (yd<11) {
        float dx = tex2D<float>(texObj, xf+1.0, yf) - tex2D<float>(texObj, xf-1.0, yf);
        float dy = tex2D<float>(texObj, xf, yf+1.0) - tex2D<float>(texObj, xf, yf-1.0);
        int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
        if (bin>31)
            bin = 0;
        float grad = sqrtf(dx*dx + dy*dy);
        atomicAdd(&hist[bin], grad*gauss[xd]*gauss[yd]);
    }
    __syncthreads();
    int x1m = (tx>=1 ? tx-1 : tx+31);
    int x1p = (tx<=30 ? tx+1 : tx-31);
    if (tx<32) {
        int x2m = (tx>=2 ? tx-2 : tx+30);
        int x2p = (tx<=29 ? tx+2 : tx-30);
        hist[tx+32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
    }
    __syncthreads();
    if (tx<32) {
        float v = hist[32+tx];
        hist[tx] = (v>hist[32+x1m] && v>=hist[32+x1p] ? v : 0.0f);
    }
    __syncthreads();
    if (tx==0) {
        float maxval1 = 0.0;
        float maxval2 = 0.0;
        int i1 = -1;
        int i2 = -1;
        for (int i=0;i<32;i++) {
            float v = hist[i];
            if (v>maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v>maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        float val1 = hist[32+((i1+1)&31)];
        float val2 = hist[32+((i1+31)&31)];
        float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
        d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);
        if (maxval2>0.8f*maxval1) {
            float val1 = hist[32+((i2+1)&31)];
            float val2 = hist[32+((i2+31)&31)];
            float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
            unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
            if (idx<d_MaxNumPoints) {
                d_Sift[idx].xpos = d_Sift[bx].xpos;
                d_Sift[idx].ypos = d_Sift[bx].ypos;
                d_Sift[idx].scale = d_Sift[bx].scale;
                d_Sift[idx].sharpness = d_Sift[bx].sharpness;
                d_Sift[idx].edgeness = d_Sift[bx].edgeness;
                d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
                d_Sift[idx].subsampling = d_Sift[bx].subsampling;
            }
        }
    }
}

// With constant number of blocks
__global__ void ComputeOrientationsCONST(hipTextureObject_t texObj, SiftPoint *d_Sift, int octave)
{
    __shared__ float hist[64];
    __shared__ float gauss[11];
    const int tx = threadIdx.x;

    int fstPts = min(d_PointCounter[2*octave-1], d_MaxNumPoints);
    int totPts = min(d_PointCounter[2*octave+0], d_MaxNumPoints);
    for (int bx = blockIdx.x + fstPts; bx < totPts; bx += gridDim.x) {

        float i2sigma2 = -1.0f/(4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
        if (tx<11)
            gauss[tx] = exp(i2sigma2*(tx-5)*(tx-5));
        if (tx<64)
            hist[tx] = 0.0f;
        __syncthreads();
        float xp = d_Sift[bx].xpos - 5.0f;
        float yp = d_Sift[bx].ypos - 5.0f;
        int yd = tx/11;
        int xd = tx - yd*11;
        float xf = xp + xd;
        float yf = yp + yd;
        if (yd<11) {
            float dx = tex2D<float>(texObj, xf+1.0, yf) - tex2D<float>(texObj, xf-1.0, yf);
            float dy = tex2D<float>(texObj, xf, yf+1.0) - tex2D<float>(texObj, xf, yf-1.0);
            int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
            if (bin>31)
                bin = 0;
            float grad = sqrtf(dx*dx + dy*dy);
            atomicAdd(&hist[bin], grad*gauss[xd]*gauss[yd]);
        }
        __syncthreads();
        int x1m = (tx>=1 ? tx-1 : tx+31);
        int x1p = (tx<=30 ? tx+1 : tx-31);
        if (tx<32) {
            int x2m = (tx>=2 ? tx-2 : tx+30);
            int x2p = (tx<=29 ? tx+2 : tx-30);
            hist[tx+32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
        }
        __syncthreads();
        if (tx<32) {
            float v = hist[32+tx];
            hist[tx] = (v>hist[32+x1m] && v>=hist[32+x1p] ? v : 0.0f);
        }
        __syncthreads();
        if (tx==0) {
            float maxval1 = 0.0;
            float maxval2 = 0.0;
            int i1 = -1;
            int i2 = -1;
            for (int i=0;i<32;i++) {
                float v = hist[i];
                if (v>maxval1) {
                    maxval2 = maxval1;
                    maxval1 = v;
                    i2 = i1;
                    i1 = i;
                } else if (v>maxval2) {
                    maxval2 = v;
                    i2 = i;
                }
            }
            float val1 = hist[32+((i1+1)&31)];
            float val2 = hist[32+((i1+31)&31)];
            float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
            d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);
            atomicMax(&d_PointCounter[2*octave+1], d_PointCounter[2*octave+0]);
            if (maxval2>0.8f*maxval1) {
                float val1 = hist[32+((i2+1)&31)];
                float val2 = hist[32+((i2+31)&31)];
                float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
                unsigned int idx = atomicInc(&d_PointCounter[2*octave+1], 0x7fffffff);
                if (idx<d_MaxNumPoints) {
                    d_Sift[idx].xpos = d_Sift[bx].xpos;
                    d_Sift[idx].ypos = d_Sift[bx].ypos;
                    d_Sift[idx].scale = d_Sift[bx].scale;
                    d_Sift[idx].sharpness = d_Sift[bx].sharpness;
                    d_Sift[idx].edgeness = d_Sift[bx].edgeness;
                    d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
                    d_Sift[idx].subsampling = d_Sift[bx].subsampling;
                }
            }
        }
        __syncthreads();
    }
}


#if 0
__global__ void OrientAndExtract(hipTextureObject_t texObj, SiftPoint *d_Sift, int fstPts, float subsampling)
{
  int totPts = min(d_PointCounter[0], d_MaxNumPoints);
  if (totPts>fstPts) {
    dim3 blocks0(totPts - fstPts);
    dim3 threads0(128);
    ComputeOrientations<<<blocks0, threads0>>>(texObj, d_Sift, fstPts);
    totPts = min(d_PointCounter[0], d_MaxNumPoints);
    dim3 blocks1(totPts - fstPts);
    dim3 threads1(16, 8);
    ExtractSiftDescriptors<<<blocks1, threads1>>>(texObj, d_Sift, fstPts, subsampling);
  }
}
#endif


///////////////////////////////////////////////////////////////////////////////
// Subtract two images (multi-scale version)
///////////////////////////////////////////////////////////////////////////////

__global__ void FindPointsMulti(float *d_Data0, SiftPoint *d_Sift, int width, int pitch, int height, float subsampling, float lowestScale, float thresh, float factor, float edgeLimit, int octave)
{
#define MEMWID (MINMAX_W + 2)
    __shared__ float ymin1[MEMWID], ymin2[MEMWID], ymin3[MEMWID];
    __shared__ float ymax1[MEMWID], ymax2[MEMWID], ymax3[MEMWID];
    __shared__ unsigned int cnt;
    __shared__ unsigned short points[96];

    int tx = threadIdx.x;
    int block = blockIdx.x/NUM_SCALES;
    int scale = blockIdx.x - NUM_SCALES*block;
    int minx = block*MINMAX_W;
    int maxx = min(minx + MINMAX_W, width);
    int xpos = minx + tx;
    int size = pitch*height;
    int ptr = size*scale + max(min(xpos-1, width-1), 0);

    if (tx==0)
        cnt = 0;
    __syncthreads();

    int yloops = min(height - MINMAX_H*blockIdx.y, MINMAX_H);
    for (int y=0;y<yloops;y++) {

        int ypos = MINMAX_H*blockIdx.y + y;
        int yptr1 = ptr + ypos*pitch;
        float maxv = fabs(d_Data0[yptr1 + 1*size]);
        maxv = fmaxf(maxv, ShiftDown(maxv, 16));
        maxv = fmaxf(maxv, ShiftDown(maxv, 8));
        maxv = fmaxf(maxv, ShiftDown(maxv, 4));
        maxv = fmaxf(maxv, ShiftDown(maxv, 2));
        maxv = fmaxf(maxv, ShiftDown(maxv, 1));
        ymax2[tx] = maxv;
        __syncthreads();
        if (fmaxf(ymax2[0], ymax2[32])<=thresh)
            continue;
        int yptr0 = ptr + max(0,ypos-1)*pitch;
        int yptr2 = ptr + min(height-1,ypos+1)*pitch;
        {
            float d10 = d_Data0[yptr0];
            float d11 = d_Data0[yptr1];
            float d12 = d_Data0[yptr2];
            ymin1[tx] = fminf(fminf(d10, d11), d12);
            ymax1[tx] = fmaxf(fmaxf(d10, d11), d12);
        }
        {
            float d30 = d_Data0[yptr0 + 2*size];
            float d31 = d_Data0[yptr1 + 2*size];
            float d32 = d_Data0[yptr2 + 2*size];
            ymin3[tx] = fminf(fminf(d30, d31), d32);
            ymax3[tx] = fmaxf(fmaxf(d30, d31), d32);
        }
        float d20 = d_Data0[yptr0 + 1*size];
        float d21 = d_Data0[yptr1 + 1*size];
        float d22 = d_Data0[yptr2 + 1*size];
        ymin2[tx] = fminf(fminf(ymin1[tx], fminf(fminf(d20, d21), d22)), ymin3[tx]);
        ymax2[tx] = fmaxf(fmaxf(ymax1[tx], fmaxf(fmaxf(d20, d21), d22)), ymax3[tx]);
        __syncthreads();
        if (tx>0 && tx<MINMAX_W+1 && xpos<=maxx) {
            if (d21<-thresh) {
                float minv = fminf(fminf(fminf(ymin2[tx-1], ymin2[tx+1]), ymin1[tx]), ymin3[tx]);
                minv = fminf(fminf(minv, d20), d22);
                if (d21<minv) {
                    int pos = atomicInc(&cnt, 31);
                    points[3*pos+0] = xpos - 1;
                    points[3*pos+1] = ypos;
                    points[3*pos+2] = scale;
                }
            }
            if (d21>thresh) {
                float maxv = fmaxf(fmaxf(fmaxf(ymax2[tx-1], ymax2[tx+1]), ymax1[tx]), ymax3[tx]);
                maxv = fmaxf(fmaxf(maxv, d20), d22);
                if (d21>maxv) {
                    int pos = atomicInc(&cnt, 31);
                    points[3*pos+0] = xpos - 1;
                    points[3*pos+1] = ypos;
                    points[3*pos+2] = scale;
                }
            }
        }
        __syncthreads();
    }
    if (tx==0)
        atomicMax(&d_PointCounter[2*octave+0], d_PointCounter[2*octave-1]);
    if (tx<cnt) {
        int xpos = points[3*tx+0];
        int ypos = points[3*tx+1];
        int scale = points[3*tx+2];
        int ptr = xpos + (ypos + (scale+1)*height)*pitch;
        float val = d_Data0[ptr];
        float *data1 = &d_Data0[ptr];
        float dxx = 2.0f*val - data1[-1] - data1[1];
        float dyy = 2.0f*val - data1[-pitch] - data1[pitch];
        float dxy = 0.25f*(data1[+pitch+1] + data1[-pitch-1] - data1[-pitch+1] - data1[+pitch-1]);
        float tra = dxx + dyy;
        float det = dxx*dyy - dxy*dxy;
        if (tra*tra<edgeLimit*det) {
            float edge = __fdividef(tra*tra, det);
            float dx = 0.5f*(data1[1] - data1[-1]);
            float dy = 0.5f*(data1[pitch] - data1[-pitch]);
            float *data0 = d_Data0 + ptr - height*pitch;
            float *data2 = d_Data0 + ptr + height*pitch;
            float ds = 0.5f*(data0[0] - data2[0]);
            float dss = 2.0f*val - data2[0] - data0[0];
            float dxs = 0.25f*(data2[1] + data0[-1] - data0[1] - data2[-1]);
            float dys = 0.25f*(data2[pitch] + data0[-pitch] - data2[-pitch] - data0[pitch]);
            float idxx = dyy*dss - dys*dys;
            float idxy = dys*dxs - dxy*dss;
            float idxs = dxy*dys - dyy*dxs;
            float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
            float idyy = dxx*dss - dxs*dxs;
            float idys = dxy*dxs - dxx*dys;
            float idss = dxx*dyy - dxy*dxy;
            float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
            float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
            float pds = idet*(idxs*dx + idys*dy + idss*ds);
            if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
                pdx = __fdividef(dx, dxx);
                pdy = __fdividef(dy, dyy);
                pds = __fdividef(ds, dss);
            }
            float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
            int maxPts = d_MaxNumPoints;
            float sc = powf(2.0f, (float)scale/NUM_SCALES) * exp2f(pds*factor);
            if (sc>=lowestScale) {
                unsigned int idx = atomicInc(&d_PointCounter[2*octave+0], 0x7fffffff);
                //d_PointCounter[2*octave+1] = d_PointCounter[2*octave+0];
                //printf("Extract %d %d %d %d\n", octave, d_PointCounter[2*octave-1], d_PointCounter[2*octave+0], d_PointCounter[2*octave+1]);
                idx = (idx>=maxPts ? maxPts-1 : idx);
                d_Sift[idx].xpos = xpos + pdx;
                d_Sift[idx].ypos = ypos + pdy;
                d_Sift[idx].scale = sc;
                d_Sift[idx].sharpness = val + dval;
                d_Sift[idx].edgeness = edge;
                d_Sift[idx].subsampling = subsampling;
            }
        }
    }
}


__global__ void LaplaceMultiTex(hipTextureObject_t texObj, float *d_Result, int width, int pitch, int height, int octave)
{
    __shared__ float data1[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
    __shared__ float data2[LAPLACE_W*LAPLACE_S];
    const int tx = threadIdx.x;
    const int xp = blockIdx.x*LAPLACE_W + tx;
    const int yp = blockIdx.y;
    const int scale = threadIdx.y;
    float *kernel = d_LaplaceKernel + octave*12*16 + scale*16;
    float *sdata1 = data1 + (LAPLACE_W + 2*LAPLACE_R)*scale;
    float x = xp-3.5;
    float y = yp+0.5;
    sdata1[tx] = kernel[4]*tex2D<float>(texObj, x, y) +
                 kernel[3]*(tex2D<float>(texObj, x, y-1.0) + tex2D<float>(texObj, x, y+1.0)) +
                 kernel[2]*(tex2D<float>(texObj, x, y-2.0) + tex2D<float>(texObj, x, y+2.0)) +
                 kernel[1]*(tex2D<float>(texObj, x, y-3.0) + tex2D<float>(texObj, x, y+3.0)) +
                 kernel[0]*(tex2D<float>(texObj, x, y-4.0) + tex2D<float>(texObj, x, y+4.0));
    __syncthreads();
    float *sdata2 = data2 + LAPLACE_W*scale;
    if (tx<LAPLACE_W) {
        sdata2[tx] = kernel[4]*sdata1[tx+4] +
                     kernel[3]*(sdata1[tx+3] + sdata1[tx+5]) +
                     kernel[2]*(sdata1[tx+2] + sdata1[tx+6]) +
                     kernel[1]*(sdata1[tx+1] + sdata1[tx+7]) +
                     kernel[0]*(sdata1[tx+0] + sdata1[tx+8]);
    }
    __syncthreads();
    if (tx<LAPLACE_W && scale<LAPLACE_S-1 && xp<width)
        d_Result[scale*height*pitch + yp*pitch + xp] = sdata2[tx] - sdata2[tx+LAPLACE_W];
}


__global__ void LaplaceMultiMemNew(float *d_Image, float *d_Result, int width, int pitch, int height, int octave)
{
    __shared__ float data1[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
    __shared__ float data2[LAPLACE_W*LAPLACE_S];
    const int tx = threadIdx.x;
    const int xp = blockIdx.x*LAPLACE_W + tx;
    const int yp = 4*blockIdx.y;
    const int scale = threadIdx.y;
    float *kernel = d_LaplaceKernel + octave*12*16 + scale*16;
    float *sdata1 = data1 + (LAPLACE_W + 2*LAPLACE_R)*scale;
    float *data = d_Image + max(min(xp - 4, width-1), 0);
    int h = height-1;
    float temp[12];
    for (int i=0;i<4;i++)
        temp[i] = data[max(0, min(yp+i-4, h))*pitch];
    for (int i=4;i<12;i++)
        temp[i] = data[min(yp+i-4, h)*pitch];
    __syncthreads();
    for (int j=0;j<4;j++) {
        sdata1[tx] = kernel[4]*temp[4+j] +
                     kernel[3]*(temp[3+j] + temp[5+j]) + kernel[2]*(temp[2+j] + temp[6+j]) +
                     kernel[1]*(temp[1+j] + temp[7+j]) + kernel[0]*(temp[0+j] + temp[8+j]);
        __syncthreads();
        float *sdata2 = data2 + LAPLACE_W*scale;
        if (tx<LAPLACE_W) {
            sdata2[tx] = kernel[4]*sdata1[tx+4] +
                         kernel[3]*(sdata1[tx+3] + sdata1[tx+5]) + kernel[2]*(sdata1[tx+2] + sdata1[tx+6]) +
                         kernel[1]*(sdata1[tx+1] + sdata1[tx+7]) + kernel[0]*(sdata1[tx+0] + sdata1[tx+8]);
        }
        __syncthreads();
        if (tx<LAPLACE_W && scale<LAPLACE_S-1 && xp<width && (yp+j)<height)
            d_Result[scale*height*pitch + (yp+j)*pitch + xp] = sdata2[tx] - sdata2[tx+LAPLACE_W];
        __syncthreads();
    }
}

__global__ void LaplaceMultiMem(float *d_Image, float *d_Result, int width, int pitch, int height, int octave)
{
    __shared__ float data1[(LAPLACE_W + 2*LAPLACE_R)*LAPLACE_S];
    __shared__ float data2[LAPLACE_W*LAPLACE_S];
    const int tx = threadIdx.x;
    const int xp = blockIdx.x*LAPLACE_W + tx;
    const int yp = blockIdx.y;
    const int scale = threadIdx.y;
    float *kernel = d_LaplaceKernel + octave*12*16 + scale*16;
    float *sdata1 = data1 + (LAPLACE_W + 2*LAPLACE_R)*scale;
    float *data = d_Image + max(min(xp - 4, width-1), 0);
    int h = height-1;
    sdata1[tx] = kernel[4]*data[min(yp, h)*pitch] +
                 kernel[3]*(data[max(0, min(yp-1, h))*pitch] + data[min(yp+1, h)*pitch]) +
                 kernel[2]*(data[max(0, min(yp-2, h))*pitch] + data[min(yp+2, h)*pitch]) +
                 kernel[1]*(data[max(0, min(yp-3, h))*pitch] + data[min(yp+3, h)*pitch]) +
                 kernel[0]*(data[max(0, min(yp-4, h))*pitch] + data[min(yp+4, h)*pitch]);
    __syncthreads();
    float *sdata2 = data2 + LAPLACE_W*scale;
    if (tx<LAPLACE_W) {
        sdata2[tx] = kernel[4]*sdata1[tx+4] +
                     kernel[3]*(sdata1[tx+3] + sdata1[tx+5]) + kernel[2]*(sdata1[tx+2] + sdata1[tx+6]) +
                     kernel[1]*(sdata1[tx+1] + sdata1[tx+7]) + kernel[0]*(sdata1[tx+0] + sdata1[tx+8]);
    }
    __syncthreads();
    if (tx<LAPLACE_W && scale<LAPLACE_S-1 && xp<width)
        d_Result[scale*height*pitch + yp*pitch + xp] = sdata2[tx] - sdata2[tx+LAPLACE_W];
}

__global__ void LowPass(float *d_Image, float *d_Result, int width, int pitch, int height)
{
    __shared__ float buffer[(LOWPASS_W + 2*LOWPASS_R)*LOWPASS_H];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int xp = blockIdx.x*LOWPASS_W + tx;
    const int yp = blockIdx.y*LOWPASS_H + ty;
    float *kernel = d_LowPassKernel;
    float *data = d_Image + max(min(xp - 4, width-1), 0);
    float *buff = buffer + ty*(LOWPASS_W + 2*LOWPASS_R);
    int h = height-1;
    if (yp<height)
        buff[tx] = kernel[4]*data[min(yp, h)*pitch] +
                   kernel[3]*(data[max(0, min(yp-1, h))*pitch] + data[min(yp+1, h)*pitch]) +
                   kernel[2]*(data[max(0, min(yp-2, h))*pitch] + data[min(yp+2, h)*pitch]) +
                   kernel[1]*(data[max(0, min(yp-3, h))*pitch] + data[min(yp+3, h)*pitch]) +
                   kernel[0]*(data[max(0, min(yp-4, h))*pitch] + data[min(yp+4, h)*pitch]);
    __syncthreads();
    if (tx<LOWPASS_W && xp<width && yp<height) {
        d_Result[yp*pitch + xp] = kernel[4]*buff[tx+4] +
                                  kernel[3]*(buff[tx+3] + buff[tx+5]) + kernel[2]*(buff[tx+2] + buff[tx+6]) +
                                  kernel[1]*(buff[tx+1] + buff[tx+7]) + kernel[0]*(buff[tx+0] + buff[tx+8]);
    }
}


////////////////////////////////////////////////////////////////////////////////////////////////


void InitCuda(int devNum)
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if (!nDevices) {
        std::cerr << "No CUDA devices available" << std::endl;
        return;
    }
    devNum = std::min(nDevices-1, devNum);
    deviceInit(devNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devNum);
    printf("Device Number: %d\n", devNum);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate/1000);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
}

float *AllocSiftTempMemory(int width, int height, int numOctaves, bool scaleUp)
{
    TimerGPU timer(0);
    const int nd = NUM_SCALES + 3;
    int w = width*(scaleUp ? 2 : 1);
    int h = height*(scaleUp ? 2 : 1);
    int p = iAlignUp(w, 128);
    int size = h*p;                 // image sizes
    int sizeTmp = nd*h*p;           // laplace buffer sizes
    for (int i=0;i<numOctaves;i++) {
        w /= 2;
        h /= 2;
        int p = iAlignUp(w, 128);
        size += h*p;
        sizeTmp += nd*h*p;
    }
    float *memoryTmp = NULL;
    size_t pitch;
    size += sizeTmp;
    safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size+4095)/4096*sizeof(float)));
#ifdef VERBOSE
    printf("Allocated memory size: %d bytes\n", size);
  printf("Memory allocation time =      %.2f ms\n\n", timer.read());
#endif
    return memoryTmp;
}

void FreeSiftTempMemory(float *memoryTmp)
{
    if (memoryTmp)
        safeCall(hipFree(memoryTmp));
}

void ExtractSift(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, bool scaleUp, float *tempMemory)
{
    TimerGPU timer(0);
    unsigned int *d_PointCounterAddr;
    safeCall(hipGetSymbolAddress((void**)&d_PointCounterAddr, d_PointCounter));
    safeCall(hipMemset(d_PointCounterAddr, 0, (8*2+1)*sizeof(int)));
    safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &siftData.maxPts, sizeof(int)));

    const int nd = NUM_SCALES + 3;
    int w = img.width*(scaleUp ? 2 : 1);
    int h = img.height*(scaleUp ? 2 : 1);
    int p = iAlignUp(w, 128);
    int width = w, height = h;
    int size = h*p;                 // image sizes
    int sizeTmp = nd*h*p;           // laplace buffer sizes
    for (int i=0;i<numOctaves;i++) {
        w /= 2;
        h /= 2;
        int p = iAlignUp(w, 128);
        size += h*p;
        sizeTmp += nd*h*p;
    }
    float *memoryTmp = tempMemory;
    size += sizeTmp;
    if (!tempMemory) {
        size_t pitch;
        safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size+4095)/4096*sizeof(float)));
#ifdef VERBOSE
        printf("Allocated memory size: %d bytes\n", size);
    printf("Memory allocation time =      %.2f ms\n\n", timer.read());
#endif
    }
    float *memorySub = memoryTmp + sizeTmp;

    CudaImage lowImg;
    lowImg.Allocate(width, height, iAlignUp(width, 128), false, memorySub);
    if (!scaleUp) {
        LowPass(lowImg, img, max(initBlur, 0.001f));
        TimerGPU timer1(0);
        float kernel[8*12*16];
        PrepareLaplaceKernels(numOctaves, 0.0f, kernel);
        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_LaplaceKernel), kernel, 8*12*16*sizeof(float)));
        int octave = ExtractSiftLoop(siftData, lowImg, numOctaves, 0.0f, thresh, lowestScale, 1.0f, memoryTmp, memorySub + height*iAlignUp(width, 128));
        safeCall(hipMemcpy(&siftData.numPts, &d_PointCounterAddr[2*octave], sizeof(int), hipMemcpyDeviceToHost));
        siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
//        printf("SIFT extraction time =        %.2f ms\n", timer1.read());
    } else {
        CudaImage upImg;
        upImg.Allocate(width, height, iAlignUp(width, 128), false, memoryTmp);
        TimerGPU timer1(0);
        ScaleUp(upImg, img);
        LowPass(lowImg, upImg, max(initBlur, 0.001f));
        float kernel[8*12*16];
        PrepareLaplaceKernels(numOctaves, 0.0f, kernel);
        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_LaplaceKernel), kernel, 8*12*16*sizeof(float)));
        int octave = ExtractSiftLoop(siftData, lowImg, numOctaves, 0.0f, thresh, lowestScale*2.0f, 1.0f, memoryTmp, memorySub + height*iAlignUp(width, 128));
        safeCall(hipMemcpy(&siftData.numPts, &d_PointCounterAddr[2*octave], sizeof(int), hipMemcpyDeviceToHost));
        siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
        RescalePositions(siftData, 0.5f);
//        printf("SIFT extraction time =        %.2f ms\n", timer1.read());
    }

    if (!tempMemory)
        safeCall(hipFree(memoryTmp));
#ifdef MANAGEDMEM
    safeCall(hipDeviceSynchronize());
#else
    if (siftData.h_data)
        safeCall(hipMemcpy(siftData.h_data, siftData.d_data, sizeof(SiftPoint)*siftData.numPts, hipMemcpyDeviceToHost));
#endif
    double totTime = timer.read();
//    printf("Incl prefiltering & memcpy =  %.2f ms %d\n\n", totTime, siftData.numPts);
}

int ExtractSiftLoop(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp, float *memorySub)
{
#ifdef VERBOSE
    TimerGPU timer(0);
#endif
    int w = img.width;
    int h = img.height;
    if (numOctaves>1) {
        CudaImage subImg;
        int p = iAlignUp(w/2, 128);
        subImg.Allocate(w/2, h/2, p, false, memorySub);
        ScaleDown(subImg, img, 0.5f);
        float totInitBlur = (float)sqrt(initBlur*initBlur + 0.5f*0.5f) / 2.0f;
        ExtractSiftLoop(siftData, subImg, numOctaves-1, totInitBlur, thresh, lowestScale, subsampling*2.0f, memoryTmp, memorySub + (h/2)*p);
    }
    if (lowestScale<subsampling*2.0f)
        ExtractSiftOctave(siftData, img, numOctaves, thresh, lowestScale, subsampling, memoryTmp);
#ifdef VERBOSE
    double totTime = timer.read();
  printf("ExtractSift time total =      %.2f ms\n\n", totTime);
#endif
    if (lowestScale<subsampling*2.0f)
        return numOctaves;
    return 0;
}

void ExtractSiftOctave(SiftData &siftData, CudaImage &img, int octave, float thresh, float lowestScale, float subsampling, float *memoryTmp)
//void ExtractSiftOctave(SiftData &siftData, CudaImage &img, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp)
{
    const int nd = NUM_SCALES + 3;
#ifdef VERBOSE
    TimerGPU timer0;
#endif
    CudaImage diffImg[nd];
    int w = img.width;
    int h = img.height;
    int p = iAlignUp(w, 128);
    for (int i=0;i<nd-1;i++)
        diffImg[i].Allocate(w, h, p, false, memoryTmp + i*p*h);

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = img.d_data;
    resDesc.res.pitch2D.width = img.width;
    resDesc.res.pitch2D.height = img.height;
    resDesc.res.pitch2D.pitchInBytes = img.pitch*sizeof(float);
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

#ifdef VERBOSE
    TimerGPU timer1;
#endif
    float baseBlur = pow(2.0f, -1.0f/NUM_SCALES);
    float diffScale = pow(2.0f, 1.0f/NUM_SCALES);
    LaplaceMulti(texObj, img, diffImg, octave);
    FindPointsMulti(diffImg, siftData, thresh, 10.0f, 1.0f/NUM_SCALES, lowestScale/subsampling, subsampling, octave);
#ifdef VERBOSE
    double gpuTimeDoG = timer1.read();
  TimerGPU timer4;
#endif
    ComputeOrientations(texObj, siftData, octave);
    ExtractSiftDescriptors(texObj, siftData, subsampling, octave);

    safeCall(hipDestroyTextureObject(texObj));
#ifdef VERBOSE
    double gpuTimeSift = timer4.read();
  double totTime = timer0.read();
  printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime-gpuTimeDoG-gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(&d_PointCounter[2]), sizeof(int)));
  int totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>0)
    printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG/NUM_SCALES, gpuTimeSift/(totPts-fstPts), totPts-fstPts);
#endif
}

void InitSiftData(SiftData &data, int num, bool host, bool dev)
{
    data.numPts = 0;
    data.maxPts = num;
    int sz = sizeof(SiftPoint)*num;
#ifdef MANAGEDMEM
    safeCall(hipMallocManaged((void **)&data.m_data, sz));
#else
    data.h_data = NULL;
    if (host)
        data.h_data = (SiftPoint *)malloc(sz);
    data.d_data = NULL;
    if (dev)
        safeCall(hipMalloc((void **)&data.d_data, sz));
#endif
}

void FreeSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
    safeCall(hipFree(data.m_data));
#else
    if (data.d_data!=NULL)
        safeCall(hipFree(data.d_data));
    data.d_data = NULL;
    if (data.h_data!=NULL)
        free(data.h_data);
#endif
    data.numPts = 0;
    data.maxPts = 0;
}

void PrintSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
    SiftPoint *h_data = data.m_data;
#else
    SiftPoint *h_data = data.h_data;
    if (data.h_data==NULL) {
        h_data = (SiftPoint *)malloc(sizeof(SiftPoint)*data.maxPts);
        safeCall(hipMemcpy(h_data, data.d_data, sizeof(SiftPoint)*data.numPts, hipMemcpyDeviceToHost));
        data.h_data = h_data;
    }
#endif
    for (int i=0;i<data.numPts;i++) {
        printf("xpos         = %.2f\n", h_data[i].xpos);
        printf("ypos         = %.2f\n", h_data[i].ypos);
        printf("scale        = %.2f\n", h_data[i].scale);
        printf("sharpness    = %.2f\n", h_data[i].sharpness);
        printf("edgeness     = %.2f\n", h_data[i].edgeness);
        printf("orientation  = %.2f\n", h_data[i].orientation);
        printf("score        = %.2f\n", h_data[i].score);
        float *siftData = (float*)&h_data[i].data;
        for (int j=0;j<8;j++) {
            if (j==0)
                printf("data = ");
            else
                printf("       ");
            for (int k=0;k<16;k++)
                if (siftData[j+8*k]<0.05)
                    printf(" .   ");
                else
                    printf("%.2f ", siftData[j+8*k]);
            printf("\n");
        }
    }
    printf("Number of available points: %d\n", data.numPts);
    printf("Number of allocated points: %d\n", data.maxPts);
}

///////////////////////////////////////////////////////////////////////////////
// Host side master functions
///////////////////////////////////////////////////////////////////////////////

double ScaleDown(CudaImage &res, CudaImage &src, float variance)
{
    static float oldVariance = -1.0f;
    if (res.d_data==NULL || src.d_data==NULL) {
        printf("ScaleDown: missing data\n");
        return 0.0;
    }
    if (oldVariance!=variance) {
        float h_Kernel[5];
        float kernelSum = 0.0f;
        for (int j=0;j<5;j++) {
            h_Kernel[j] = (float)expf(-(double)(j-2)*(j-2)/2.0/variance);
            kernelSum += h_Kernel[j];
        }
        for (int j=0;j<5;j++)
            h_Kernel[j] /= kernelSum;
        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_ScaleDownKernel), h_Kernel, 5*sizeof(float)));
        oldVariance = variance;
    }
    dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
    dim3 threads(SCALEDOWN_W + 4);
    ScaleDown<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch);
    checkMsg("ScaleDown() execution failed\n");
    return 0.0;
}

double ScaleUp(CudaImage &res, CudaImage &src)
{
    if (res.d_data==NULL || src.d_data==NULL) {
        printf("ScaleUp: missing data\n");
        return 0.0;
    }
    dim3 blocks(iDivUp(res.width, SCALEUP_W), iDivUp(res.height, SCALEUP_H));
    dim3 threads(SCALEUP_W, SCALEUP_H);
    ScaleUp<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch);
    checkMsg("ScaleUp() execution failed\n");
    return 0.0;
}

double ComputeOrientations(hipTextureObject_t texObj, SiftData &siftData, int octave)
{
    dim3 blocks(256);
    dim3 threads(128);
#ifdef MANAGEDMEM
    ComputeOrientationsCONST<<<blocks, threads>>>(texObj, siftData.m_data, octave);
#else
    ComputeOrientationsCONST<<<blocks, threads>>>(texObj, siftData.d_data, octave);
#endif
    checkMsg("ComputeOrientations() execution failed\n");
    return 0.0;
}

double ExtractSiftDescriptors(hipTextureObject_t texObj, SiftData &siftData,float subsampling, int octave)
{
    dim3 blocks(256);
    dim3 threads(16, 8);
#ifdef MANAGEDMEM
    ExtractSiftDescriptorsCONST<<<blocks, threads>>>(texObj, siftData.m_data, subsampling, octave);
#else
    ExtractSiftDescriptorsCONST<<<blocks, threads>>>(texObj, siftData.d_data, subsampling, octave);
#endif
    checkMsg("ExtractSiftDescriptors() execution failed\n");
    return 0.0;
}

#if 0
double OrientAndExtract(hipTextureObject_t texObj, SiftData &siftData, int fstPts, float subsampling)
{
#ifdef MANAGEDMEM
  OrientAndExtract<<<1,1>>>(texObj, siftData.m_data, fstPts, subsampling);
#else
  OrientAndExtract<<<1,1>>>(texObj, siftData.d_data, fstPts, subsampling);
#endif
  checkMsg("OrientAndExtract() execution failed\n");
  return 0.0;
}
#endif

double RescalePositions(SiftData &siftData, float scale)
{
    dim3 blocks(iDivUp(siftData.numPts, 64));
    dim3 threads(64);
    RescalePositions<<<blocks, threads>>>(siftData.d_data, siftData.numPts, scale);
    checkMsg("RescapePositions() execution failed\n");
    return 0.0;
}

double LowPass(CudaImage &res, CudaImage &src, float scale)
{
    float kernel[2*LOWPASS_R+1];
    static float oldScale = -1.0f;
    if (scale!=oldScale) {
        float kernelSum = 0.0f;
        float ivar2 = 1.0f/(2.0f*scale*scale);
        for (int j=-LOWPASS_R;j<=LOWPASS_R;j++) {
            kernel[j+LOWPASS_R] = (float)expf(-(double)j*j*ivar2);
            kernelSum += kernel[j+LOWPASS_R];
        }
        for (int j=-LOWPASS_R;j<=LOWPASS_R;j++)
            kernel[j+LOWPASS_R] /= kernelSum;
        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_LowPassKernel), kernel, (2*LOWPASS_R+1)*sizeof(float)));
        oldScale = scale;
    }
    int width = res.width;
    int pitch = res.pitch;
    int height = res.height;
    dim3 blocks(iDivUp(width, LOWPASS_W), iDivUp(height, LOWPASS_H));
    dim3 threads(LOWPASS_W+2*LOWPASS_R, LOWPASS_H);
    LowPass<<<blocks, threads>>>(src.d_data, res.d_data, width, pitch, height);
    checkMsg("LowPass() execution failed\n");
    return 0.0;
}

//==================== Multi-scale functions ===================//

void PrepareLaplaceKernels(int numOctaves, float initBlur, float *kernel)
{
    if (numOctaves>1) {
        float totInitBlur = (float)sqrt(initBlur*initBlur + 0.5f*0.5f) / 2.0f;
        PrepareLaplaceKernels(numOctaves-1, totInitBlur, kernel);
    }
    float scale = pow(2.0f, -1.0f/NUM_SCALES);
    float diffScale = pow(2.0f, 1.0f/NUM_SCALES);
    for (int i=0;i<NUM_SCALES+3;i++) {
        float kernelSum = 0.0f;
        float var = scale*scale - initBlur*initBlur;
        for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) {
            kernel[numOctaves*12*16 + 16*i+j+LAPLACE_R] = (float)expf(-(double)j*j/2.0/var);
            kernelSum += kernel[numOctaves*12*16 + 16*i+j+LAPLACE_R];
        }
        for (int j=-LAPLACE_R;j<=LAPLACE_R;j++)
            kernel[numOctaves*12*16 + 16*i+j+LAPLACE_R] /= kernelSum;
        scale *= diffScale;
    }
}

double LaplaceMulti(hipTextureObject_t texObj, CudaImage &baseImage, CudaImage *results, int octave)
{
    int width = results[0].width;
    int pitch = results[0].pitch;
    int height = results[0].height;
    dim3 threads(LAPLACE_W+2*LAPLACE_R, LAPLACE_S);
#if 1
    dim3 blocks(iDivUp(width, LAPLACE_W), iDivUp(height, 4));
    LaplaceMultiMemNew<<<blocks, threads>>>(baseImage.d_data, results[0].d_data, width, pitch, height, octave);
#else
    dim3 blocks(iDivUp(width, LAPLACE_W), height);
  LaplaceMultiTex<<<blocks, threads>>>(texObj, results[0].d_data, width, pitch, height, octave);
#endif
    checkMsg("LaplaceMulti() execution failed\n");
    return 0.0;
}

double FindPointsMulti(CudaImage *sources, SiftData &siftData, float thresh, float edgeLimit, float factor, float lowestScale, float subsampling, int octave)
{
    if (sources->d_data==NULL) {
        printf("FindPointsMulti: missing data\n");
        return 0.0;
    }
    int w = sources->width;
    int p = sources->pitch;
    int h = sources->height;
    dim3 blocks(iDivUp(w, MINMAX_W)*NUM_SCALES, iDivUp(h, MINMAX_H));
    dim3 threads(MINMAX_W + 2);
#ifdef MANAGEDMEM
    FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.m_data, w, p, h, subsampling, lowestScale, thresh, factor, edgeLimit, octave);
#else
    FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.d_data, w, p, h, subsampling, lowestScale, thresh, factor, edgeLimit, octave);
#endif
    checkMsg("FindPointsMulti() execution failed\n");
    return 0.0;
}


////////////////////////////////////////////////////////////////////////////////////////////////


SiftData::SiftData(int num)
        : h_data(NULL),
          d_data(NULL)
{
    InitSiftData(*this, num, true, true);
}

SiftData::SiftData(const SiftData &data)
        : h_data(NULL),
          d_data(NULL)
{
    InitSiftData(*this, data.maxPts, true, true);
    numPts = data.numPts;
    if (h_data != NULL && data.h_data != NULL) std::memcpy(h_data, data.h_data, numPts*sizeof(SiftPoint));
    if (d_data != NULL && data.d_data != NULL) safeCall(hipMemcpy(d_data, data.d_data, numPts*sizeof(SiftPoint), hipMemcpyDeviceToDevice));
}

SiftData::~SiftData() {
    FreeSiftData(*this);
}

SiftData& SiftData::operator=(const SiftData &data) {
    if (this != &data) {
        InitSiftData(*this, data.maxPts, true, true);
        numPts = data.numPts;
        if (h_data != NULL && data.h_data != NULL) std::memcpy(h_data, data.h_data, numPts*sizeof(SiftPoint));
        if (d_data != NULL && data.d_data != NULL) safeCall(hipMemcpy(d_data, data.d_data, numPts*sizeof(SiftPoint), hipMemcpyDeviceToDevice));
    }
    return *this;
}

void SiftData::resize(size_t new_size) {
    if (new_size <= (size_t)maxPts) {
        numPts = (int)new_size;
        return;
    }
    reserve(new_size);  // Conservative?
    numPts = (int)new_size;
}

void SiftData::reserve(size_t new_capacity) {
    if (new_capacity <= (size_t)maxPts) return;

    size_t mem_sz = new_capacity*sizeof(SiftPoint);

    if (h_data != NULL) {
        SiftPoint * h_data_old = h_data;
        h_data = (SiftPoint *)malloc(mem_sz);
        std::memcpy(h_data, h_data_old, numPts*sizeof(SiftPoint));
        free(h_data_old);
    } else {
        h_data = (SiftPoint *)malloc(mem_sz);
    }

    if (d_data != NULL) {
        SiftPoint * d_data_old = d_data;
        safeCall(hipMalloc((void **)&d_data, mem_sz));
        safeCall(hipMemcpy(d_data, d_data_old, numPts*sizeof(SiftPoint), hipMemcpyDeviceToDevice));
        safeCall(hipFree(d_data_old));
    } else {
        safeCall(hipMalloc((void **)&d_data, mem_sz));
    }

    maxPts = (int)new_capacity;
}

void SiftData::freeBuffers() {
    FreeSiftData(*this);
}

SiftData& SiftData::append(const SiftData &data) {
    int new_sz = numPts + data.numPts;
    reserve((size_t)(new_sz));
    size_t cp_sz = data.numPts*sizeof(SiftPoint);
    if (h_data != NULL && data.h_data != NULL) std::memcpy(&h_data[numPts], data.h_data, cp_sz);
    if (d_data != NULL && data.d_data != NULL) safeCall(hipMemcpy(&d_data[numPts], data.d_data, cp_sz, hipMemcpyDeviceToDevice));
    numPts = new_sz;
    return *this;
}

void SiftData::syncHostToDevice() {
    if (h_data != NULL && d_data != NULL) safeCall(hipMemcpy(d_data, h_data, numPts*sizeof(SiftPoint), hipMemcpyHostToDevice));
}

void SiftData::syncDeviceToHost() {
    if (h_data != NULL && d_data != NULL) safeCall(hipMemcpy(h_data, d_data, numPts*sizeof(SiftPoint), hipMemcpyDeviceToHost));
}
